#include "hip/hip_runtime.h"
/*
ncu --csv --log-file build/bench.csv --cache-control=all --clock-control=base --metrics gpu__time_duration.sum ./build/bench_gemm 4096 4096 4096
ncu --csv --log-file build/bench_bank_conflicts.csv --metrics  l1tex__data_bank_conflicts_pipe_lsu_mem_shared,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum ./build/bench_gemm 4096 4096 4096

ncu -f --set full -o build/bench_gemm_report ./build/bench_gemm 4096 4096 4096
*/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <fstream>
#include "cutlass_gemm_v2.h"

#include "kittens_gemm.h"
#include "kittens_gemm_v1.h"
#include "kittens_gemm_v2.h"
#include "kittens_gemm_v3.h"
#include "kittens_gemm_128.h"
#include "blas.h"


float frand() {
    return (float)rand() / (float)RAND_MAX;
}

void generate_tensor(float * tensor, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        tensor[i] = frand();
    }
}

void generate_range_tensor(float * tensor, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        tensor[i] = i;
    }
}

void print_tensor(half * tensor, int M, int N) {
    printf("[");
    for (int i = 0; i < M; i++) {
        printf("[");
        for (int j = 0; j < N; j++) {
            int offset = i * N + j;
            printf("%.5f, ", __half2float(tensor[offset]));

        }
        printf("],\n");
    }
    printf("]\n");
}

bool check_value(float abs_tol, float rel_tol, half *h_d_c, half *h_c, int m, int n) {
    for (size_t i = 0; i < m; i++) {
        for (size_t j = 0; j < n; j++) {
            float gpu_value = (float)h_d_c[i * n + j];
            float cpu_value = (float)h_c[i * n + j];
            float diff = abs(gpu_value - cpu_value);
            if (diff > max(abs_tol, cpu_value * rel_tol)) {
                std::cout << "blas[" << i << ", " << j << "] = " << gpu_value 
                << ", manual[" << i << ", " << j << "] = " << cpu_value
                << " Abs Diff: " << diff << std::endl;
                return false;
            }
        }
    }
    return true;
}

void bench_helper(int M, int N, int K) {

}
// int equal_tensor(float * tensor1, float * tensor2, int N, int C, int H, int W) {
//     int equal = 1;
//     for (int n = 0; n < N; n++) {
//         for (int c = 0; c < C; c++) {
//             for (int h = 0; h < H; h++) {
//                 for (int w = 0; w < W; w++) {
//                     int offset = n * C * H * W + c * H * W + h * W + w;
//                     if (tensor1[offset] != tensor2[offset]) {
//                         printf("tensor1[%d, %d, %d, %d] = %.3f, tensor2[%d, %d, %d, %d] = %.3f\n", 
//                                n, c, h, w, tensor1[offset], n, c, h, w, tensor2[offset]);
//                         equal = 0;
//                     }
//                 }
//             }
//         }
//     }
//     return equal;
// }

int main(int argc, char ** argv) {
    srand(0);

    std::ofstream outFile("build/bench_gemm.csv");
    outFile << "matrix size(N),cublas_gemm,cutlass_gemm_v2,kittens_gemm,kittens_gemm_v1,kittens_gemm_v2,kittens_gemm_v3\n";
    std::vector<std::vector<int>> prob_shape;
    for (int i = 1; i <= 64; i++) {
        prob_shape.push_back({256 * i, 256 * i, 256 * i});
    }

    for (auto &shape : prob_shape) {
        int M = shape[0];
        int N = shape[1];
        int K = shape[2];
        std::cout << "M: " << M << ", N: " << N << ", K: " << K << std::endl;
        outFile << M << "," << N << "," << K << ",";
        thrust::host_vector<half> h_A(M * K);
        thrust::host_vector<half> h_B(N * K);
        thrust::host_vector<half> h_C(M * N);
        thrust::host_vector<half> h_C1(M * N);

        for (int j = 0; j < M * K; ++j) h_A[j] = static_cast<half>( 2*(rand() / double(RAND_MAX)) - 1 );
        for (int j = 0; j < N * K; ++j) h_B[j] = static_cast<half>( 2*(rand() / double(RAND_MAX)) - 1 );
        for (int j = 0; j < M * N; ++j) h_C[j] = static_cast<half>(-1);
        for (int j = 0; j < M * N; ++j) h_C1[j] = static_cast<half>(-1);

        thrust::device_vector<half> d_A = h_A;
        thrust::device_vector<half> d_B = h_B;
        thrust::device_vector<half> d_C = h_C;
        double gflops = (2.0*M*N*K) * 1e-9;

        int repeat = 10;
        double used_time = 0.0;

        constexpr float abs_tol = 1.0e-0f;
        constexpr float rel_tol = 1.0e-0f;

        thrust::fill(d_C.begin(), d_C.end(), 0.0);
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        blas_matmul<half>(&handle, d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        thrust::copy(d_C.begin(), d_C.end(), h_C.begin());
        // print_tensor(h_C.data(), M, N);
        for (int i = 0; i < repeat; i++) {
            thrust::fill(d_C.begin(), d_C.end(), 0.0);
            used_time += blas_matmul<half>(&handle, d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        }
        used_time /= repeat;
        std::cout << "cublas_gemm MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, TFLOPS: "
                << gflops/used_time << std::endl;
        hipblasDestroy(handle);
        outFile << gflops/used_time << ",";

        thrust::fill(d_C.begin(), d_C.end(), 0.0);
        bench::cutlass_gemm_v2::cutlass_gemm<half>(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        thrust::copy(d_C.begin(), d_C.end(), h_C1.begin());
        // print_tensor(h_C1.data(), M, N);
        used_time = 0.0;
        for (int i = 0; i < repeat; i++) {
            thrust::fill(d_C.begin(), d_C.end(), 0.0);
            used_time += bench::cutlass_gemm_v2::cutlass_gemm<half>(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        }
        used_time /= repeat;
        std::cout << "cutlass_gemm_v2 MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, TFLOPS: "
                << gflops/used_time << std::endl;
        outFile << gflops/used_time << ",";
        if (check_value(abs_tol, rel_tol, h_C.data(), h_C1.data(), M, N)) {
            std::cout << "Test PASSED" << std::endl;
        } else {
            std::cout << "Test FAILED" << std::endl;
        }

        // thrust::fill(d_C.begin(), d_C.end(), 0.0);
        // bench::kittens_gemm::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // thrust::copy(d_C.begin(), d_C.end(), h_C1.begin());
        // // print_tensor(h_C1.data(), M, N);
        // used_time = 0.0;
        // for (int i = 0; i < repeat; i++) {
        //     thrust::fill(d_C.begin(), d_C.end(), 0.0);
        //     used_time += bench::kittens_gemm::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // }
        // used_time /= repeat;
        // std::cout << "kittens_gemm MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, GFLOPS: "
        //         << gflops/used_time << std::endl;
        // outFile << gflops/used_time << ",";
        // if (check_value(abs_tol, rel_tol, h_C.data(), h_C1.data(), M, N)) {
        //     std::cout << "Test PASSED" << std::endl;
        // } else {
        //     std::cout << "Test FAILED" << std::endl;
        // }

        // thrust::fill(d_C.begin(), d_C.end(), 0.0);
        // bench::kittens_gemm_v1::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // thrust::copy(d_C.begin(), d_C.end(), h_C1.begin());
        // // print_tensor(h_C1.data(), M, N);
        // used_time = 0.0;
        // for (int i = 0; i < repeat; i++) {
        //     thrust::fill(d_C.begin(), d_C.end(), 0.0);
        //     used_time += bench::kittens_gemm_v1::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // }
        // used_time /= repeat;
        // std::cout << "kittens_gemm_v1 MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, GFLOPS: "
        //         << gflops/used_time << std::endl;
        // outFile << gflops/used_time << ",";
        // if (check_value(abs_tol, rel_tol, h_C.data(), h_C1.data(), M, N)) {
        //     std::cout << "Test PASSED" << std::endl;
        // } else {
        //     std::cout << "Test FAILED" << std::endl;
        // }

        // thrust::fill(d_C.begin(), d_C.end(), 0.0);
        // bench::kittens_gemm_v2::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // thrust::copy(d_C.begin(), d_C.end(), h_C1.begin());
        // // print_tensor(h_C1.data(), M, N);
        // used_time = 0.0;
        // for (int i = 0; i < repeat; i++) {
        //     thrust::fill(d_C.begin(), d_C.end(), 0.0);
        //     used_time += bench::kittens_gemm_v2::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // }
        // used_time /= repeat;
        // std::cout << "kittens_gemm_v2 MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, GFLOPS: "
        //         << gflops/used_time << std::endl;
        // outFile << gflops/used_time << ",";
        // if (check_value(abs_tol, rel_tol, h_C.data(), h_C1.data(), M, N)) {
        //     std::cout << "Test PASSED" << std::endl;
        // } else {
        //     std::cout << "Test FAILED" << std::endl;
        // }

        // thrust::fill(d_C.begin(), d_C.end(), 0.0);
        // bench::kittens_gemm_v3::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // thrust::copy(d_C.begin(), d_C.end(), h_C1.begin());
        // // print_tensor(h_C1.data(), M, N);
        // used_time = 0.0;
        // for (int i = 0; i < repeat; i++) {
        //     thrust::fill(d_C.begin(), d_C.end(), 0.0);
        //     used_time += bench::kittens_gemm_v3::kittens_gemm(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
        // }
        // used_time /= repeat;
        // std::cout << "kittens_gemm_v3 MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, GFLOPS: "
        //         << gflops/used_time << std::endl;
        // outFile << gflops/used_time << ",";

        // if (check_value(abs_tol, rel_tol, h_C.data(), h_C1.data(), M, N)) {
        //     std::cout << "Test PASSED" << std::endl;
        // } else {
        //     std::cout << "Test FAILED" << std::endl;
        // }

        outFile << "\n";
    }



    return 0;
}