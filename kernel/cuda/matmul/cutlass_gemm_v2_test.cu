#include "hip/hip_runtime.h"
/*
ncu --csv --log-file build/bench.csv --cache-control=all --clock-control=base --metrics gpu__time_duration.sum ./build/bench_gemm 4096 4096 4096
ncu --csv --log-file build/bench_bank_conflicts.csv --metrics  l1tex__data_bank_conflicts_pipe_lsu_mem_shared,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum ./build/bench_gemm 4096 4096 4096

ncu -f --set full -o build/bench_gemm_report ./build/bench_gemm 4096 4096 4096
*/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "cutlass_gemm_v2.h"
#include "blas.h"


float frand() {
    return (float)rand() / (float)RAND_MAX;
}

void generate_tensor(float * tensor, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        tensor[i] = frand();
    }
}

void generate_range_tensor(float * tensor, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        tensor[i] = i;
    }
}

void print_tensor(half * tensor, int M, int N) {
    printf("[");
    for (int i = 0; i < M; i++) {
        printf("[");
        for (int j = 0; j < N; j++) {
            int offset = i * N + j;
            printf("%.5f, ", __half2float(tensor[offset]));

        }
        printf("],\n");
    }
    printf("]\n");
}

bool check_value(float abs_tol, float rel_tol, half *h_d_c, half *h_c, int m, int n) {
    for (size_t i = 0; i < m; i++) {
        for (size_t j = 0; j < n; j++) {
            float gpu_value = (float)h_d_c[i * n + j];
            float cpu_value = (float)h_c[i * n + j];
            float diff = abs(gpu_value - cpu_value);
            if (diff > max(abs_tol, cpu_value * rel_tol)) {
                std::cout << "blas[" << i << ", " << j << "] = " << gpu_value 
                << ", manual[" << i << ", " << j << "] = " << cpu_value
                << " Abs Diff: " << diff << std::endl;
                return false;
            }
        }
    }
    return true;
}

int main(int argc, char ** argv) {
    srand(0);
    
    int M = 2227200;
    if (argc >= 2) {
        sscanf(argv[1], "%d", &M);
    }
    int N = 128;
    if (argc >= 3) {
        sscanf(argv[2], "%d", &N);
    }
    int K = 128 * 3;
    if (argc >= 4) {
        sscanf(argv[3], "%d", &K);
    }

    std::cout << "M: " << M << ", N: " << N << ", K: " << K << std::endl;

    thrust::host_vector<half> h_A(M * K);
    thrust::host_vector<half> h_B(N * K);
    thrust::host_vector<half> h_C(M * N);
    thrust::host_vector<half> h_C1(M * N);

    for (int j = 0; j < M * K; ++j) h_A[j] = static_cast<half>( 2*(rand() / double(RAND_MAX)) - 1 );
    for (int j = 0; j < N * K; ++j) h_B[j] = static_cast<half>( 2*(rand() / double(RAND_MAX)) - 1 );
    for (int j = 0; j < M * N; ++j) h_C[j] = static_cast<half>(-1);
    for (int j = 0; j < M * N; ++j) h_C1[j] = static_cast<half>(-1);

    thrust::device_vector<half> d_A = h_A;
    thrust::device_vector<half> d_B = h_B;
    thrust::device_vector<half> d_C = h_C;
    double gflops = (2.0*M*N*K) * 1e-9;

    constexpr float abs_tol = 1.0e-0f;
    constexpr float rel_tol = 1.0e-0f;

    thrust::fill(d_C.begin(), d_C.end(), 0.0);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    blas_matmul<half>(&handle, d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
    thrust::copy(d_C.begin(), d_C.end(), h_C.begin());
    // print_tensor(h_C.data(), M, N)
    hipblasDestroy(handle);

    thrust::fill(d_C.begin(), d_C.end(), 0.0);
    bench::cutlass_gemm_v2::cutlass_gemm<half>(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
    thrust::copy(d_C.begin(), d_C.end(), h_C1.begin());
    // print_tensor(h_C1.data(), M, N);

    if (check_value(abs_tol, rel_tol, h_C.data(), h_C1.data(), M, N)) {
        std::cout << "Test PASSED" << std::endl;
    } else {
        std::cout << "Test FAILED" << std::endl;
    }
    return 0;
}