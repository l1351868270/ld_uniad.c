#include "hip/hip_runtime.h"
/*
make cudnn_frontend_gemm_test
ncu --csv --log-file build/bench.csv --cache-control=all --clock-control=base --metrics gpu__time_duration.sum ./build/cudnn_frontend_gemm_test 4096 4096 4096
ncu --csv --log-file build/bench_bank_conflicts.csv --metrics  l1tex__data_bank_conflicts_pipe_lsu_mem_shared,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum ./build/cudnn_frontend_gemm_test 4096 4096 4096

ncu -f --set full -o build/cutlass_gemm_v2_test_report ./build/cudnn_frontend_gemm_test 4096 4096 4096
*/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "cudnn_frontend_gemm.h"
#include "blas.h"


float frand() {
    return (float)rand() / (float)RAND_MAX;
}

void generate_tensor(float * tensor, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        tensor[i] = frand();
    }
}

void generate_range_tensor(float * tensor, int M, int N) {
    for (int i = 0; i < M * N; i++) {
        tensor[i] = i;
    }
}

void print_tensor(half * tensor, int M, int N) {
    printf("[");
    for (int i = 0; i < M; i++) {
        printf("[");
        for (int j = 0; j < N; j++) {
            int offset = i * N + j;
            printf("%.5f, ", __half2float(tensor[offset]));

        }
        printf("],\n");
    }
    printf("]\n");
}

bool check_value(float abs_tol, float rel_tol, half *h_d_c, half *h_c, int m, int n) {
    for (size_t i = 0; i < m; i++) {
        for (size_t j = 0; j < n; j++) {
            float gpu_value = (float)h_d_c[i * n + j];
            float cpu_value = (float)h_c[i * n + j];
            float diff = abs(gpu_value - cpu_value);
            if (diff > max(abs_tol, cpu_value * rel_tol)) {
                std::cout << "blas[" << i << ", " << j << "] = " << gpu_value 
                << ", manual[" << i << ", " << j << "] = " << cpu_value
                << " Abs Diff: " << diff << std::endl;
                return false;
            }
        }
    }
    return true;
}

int main(int argc, char ** argv) {
    srand(0);
    
    int M = 2227200;
    if (argc >= 2) {
        sscanf(argv[1], "%d", &M);
    }
    int N = 128;
    if (argc >= 3) {
        sscanf(argv[2], "%d", &N);
    }
    int K = 128 * 3;
    if (argc >= 4) {
        sscanf(argv[3], "%d", &K);
    }

    std::cout << "M: " << M << ", N: " << N << ", K: " << K << std::endl;

    thrust::host_vector<half> h_A(M * K);
    thrust::host_vector<half> h_B(N * K);
    thrust::host_vector<half> h_C(M * N);
    thrust::host_vector<half> h_C1(M * N);

    for (int j = 0; j < M * K; ++j) h_A[j] = static_cast<half>( 2*(rand() / double(RAND_MAX)) - 1 );
    for (int j = 0; j < N * K; ++j) h_B[j] = static_cast<half>( 2*(rand() / double(RAND_MAX)) - 1 );
    for (int j = 0; j < M * N; ++j) h_C[j] = static_cast<half>(-1);
    for (int j = 0; j < M * N; ++j) h_C1[j] = static_cast<half>(-1);

    thrust::device_vector<half> d_A = h_A;
    thrust::device_vector<half> d_B = h_B;
    thrust::device_vector<half> d_C = h_C;
    double gflops = (2.0*M*N*K) * 1e-9;

    constexpr float abs_tol = 1.0e-0f;
    constexpr float rel_tol = 1.0e-0f;

    double used_time = 0.0;
    int repeat = 10;

    thrust::fill(d_C.begin(), d_C.end(), 0.0);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    blas_matmul<half>(&handle, d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
    thrust::copy(d_C.begin(), d_C.end(), h_C.begin());
    // print_tensor(h_C.data(), M, N);
    for (int i = 0; i < repeat; i++) {
        thrust::fill(d_C.begin(), d_C.end(), 0.0);
        used_time += blas_matmul<half>(&handle, d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
    }
    used_time /= repeat;
    std::cout << "cublas_gemm MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, GFLOPS: "
            << gflops/used_time << std::endl;
    hipblasDestroy(handle);

    sleep(1);  
    thrust::fill(d_C.begin(), d_C.end(), 0.0);
    bench::cudnn_frontend_gemm::cudnn_gemm<half>(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
    thrust::copy(d_C.begin(), d_C.end(), h_C1.begin());
    // print_tensor(h_C1.data(), M, N);
    used_time = 0.0;
    for (int i = 0; i < repeat; i++) {
        thrust::fill(d_C.begin(), d_C.end(), 0.0);
        used_time += bench::cudnn_frontend_gemm::cudnn_gemm<half>(d_C.data().get(), d_A.data().get(), d_B.data().get(), M, N, K);
    }
    used_time /= repeat;
    std::cout << "cutlass_gemm_v2 MNK:" << M << "*" << N << "*" << K << ", GFLOPs:" << gflops <<", used_time: " << used_time << "ms, GFLOPS: "
            << gflops/used_time << std::endl;

    if (check_value(abs_tol, rel_tol, h_C.data(), h_C1.data(), M, N)) {
        std::cout << "Test PASSED" << std::endl;
    } else {
        std::cout << "Test FAILED" << std::endl;
    }
    return 0;
}